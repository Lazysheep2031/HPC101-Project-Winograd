#include "hip/hip_runtime.h"
#include "winograd.cuh"

// F(2x2, 3x3) Winograd算法的变换矩阵
__constant__ float G[4][3] = {
    {1.0f, 0.0f, 0.0f}, 
    {0.5f, 0.5f, 0.5f}, 
    {0.5f, -0.5f, 0.5f}, 
    {0.0f, 0.0f, 1.0f}
};

__constant__ float B_T[4][4] = {
    {1.0f, 0.0f, -1.0f, 0.0f}, 
    {0.0f, 1.0f, 1.0f, 0.0f}, 
    {0.0f, -1.0f, 1.0f, 0.0f}, 
    {0.0f, 1.0f, 0.0f, -1.0f}
};

__constant__ float B[4][4] = {
    {1.0f,  0.0f,  0.0f,  0.0f}, 
    {0.0f,  1.0f, -1.0f,  1.0f}, 
    {-1.0f, 1.0f,  1.0f,  0.0f}, 
    {0.0f,  0.0f,  0.0f, -1.0f}
};

__constant__ float A_T[2][4] = {
    {1.0f, 1.0f, 1.0f, 0.0f}, 
    {0.0f, 1.0f, -1.0f, -1.0f}
};

// Step 5: 向量化内存访问优化的Winograd卷积核函数
__global__
void winograd_conv_kernel(const float* __restrict__ image,
                          const float* __restrict__ filter,
                          float* __restrict__ output,
                          int N, int C, int H, int W, int K, int outH, int outW) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int num_tiles = N * K * (outH / 2) * (outW / 2);
    if (idx >= num_tiles) return;

    // 将线程索引分解为(n, k, tile_y, tile_x)
    int p_local = idx % ((outH / 2) * (outW / 2));
    int k = (idx / ((outH / 2) * (outW / 2))) % K;
    int n = idx / (K * (outH / 2) * (outW / 2));
    int tile_y = p_local / (outW / 2);
    int tile_x = p_local % (outW / 2);

    float m[4][4] = {{0.0f}};

    // 遍历输入通道
    for (int c = 0; c < C; ++c) {
        // === 使用向量化访问优化的滤波器变换 ===
        const float* g = filter + (k * C + c) * 9;
        float u_kc[4][4];
        
        // 将滤波器权重加载到寄存器中
        float g0 = g[0], g1 = g[1], g2 = g[2];
        float g3 = g[3], g4 = g[4], g5 = g[5];
        float g6 = g[6], g7 = g[7], g8 = g[8];
        
        // 预计算公共项
        float t1 = g0 + g3 + g6;
        float t2 = g1 + g4 + g7;
        float t3 = g2 + g5 + g8;
        float t4 = g0 - g3 + g6;
        float t5 = g1 - g4 + g7;
        float t6 = g2 - g5 + g8;
        
        // 行0
        u_kc[0][0] = g0;
        u_kc[0][1] = 0.5f * (g0 + g1 + g2);
        u_kc[0][2] = 0.5f * (g0 - g1 + g2);
        u_kc[0][3] = g2;
        
        // 行1
        u_kc[1][0] = 0.5f * t1;
        u_kc[1][1] = 0.25f * (t1 + t2 + t3);
        u_kc[1][2] = 0.25f * (t1 - t2 + t3);
        u_kc[1][3] = 0.5f * t3;
        
        // 行2
        u_kc[2][0] = 0.5f * t4;
        u_kc[2][1] = 0.25f * (t4 + t5 + t6);
        u_kc[2][2] = 0.25f * (t4 - t5 + t6);
        u_kc[2][3] = 0.5f * t6;
        
        // 行3
        u_kc[3][0] = g6;
        u_kc[3][1] = 0.5f * (g6 + g7 + g8);
        u_kc[3][2] = 0.5f * (g6 - g7 + g8);
        u_kc[3][3] = g8;

        // === 使用向量化内存访问优化的图像变换 ===
        int h_start = tile_y * 2;
        int w_start = tile_x * 2;
        float d[4][4];
        
        // 在可能的情况下使用向量化内存访问
        int base_idx = (n * C + c) * H * W + h_start * W + w_start;
        float4* image_ptr = (float4*)image;
        
        // 检查是否可以使用向量化访问（4元素对齐）
        if (W % 4 == 0 && w_start % 4 == 0) {
            // 每行的向量化加载
            float4 row0 = image_ptr[base_idx / 4];
            float4 row1 = image_ptr[(base_idx + W) / 4];
            float4 row2 = image_ptr[(base_idx + 2*W) / 4];
            float4 row3 = image_ptr[(base_idx + 3*W) / 4];
            
            d[0][0] = row0.x; d[0][1] = row0.y; d[0][2] = row0.z; d[0][3] = row0.w;
            d[1][0] = row1.x; d[1][1] = row1.y; d[1][2] = row1.z; d[1][3] = row1.w;
            d[2][0] = row2.x; d[2][1] = row2.y; d[2][2] = row2.z; d[2][3] = row2.w;
            d[3][0] = row3.x; d[3][1] = row3.y; d[3][2] = row3.z; d[3][3] = row3.w;
        } else {
            // 回退到常规访问
            d[0][0] = image[base_idx];
            d[0][1] = image[base_idx + 1];
            d[0][2] = image[base_idx + 2];
            d[0][3] = image[base_idx + 3];
            d[1][0] = image[base_idx + W];
            d[1][1] = image[base_idx + W + 1];
            d[1][2] = image[base_idx + W + 2];
            d[1][3] = image[base_idx + W + 3];
            d[2][0] = image[base_idx + 2*W];
            d[2][1] = image[base_idx + 2*W + 1];
            d[2][2] = image[base_idx + 2*W + 2];
            d[2][3] = image[base_idx + 2*W + 3];
            d[3][0] = image[base_idx + 3*W];
            d[3][1] = image[base_idx + 3*W + 1];
            d[3][2] = image[base_idx + 3*W + 2];
            d[3][3] = image[base_idx + 3*W + 3];
        }
        
        float v_ncp[4][4];
        
        // 直接计算 B^T * d * B（避免中间临时数组temp_d）
        // 使用优化的直接公式计算 B^T * d * B
        v_ncp[0][0] = d[0][0] - d[0][2] - d[2][0] + d[2][2];
        v_ncp[0][1] = d[0][1] + d[0][2] - d[2][1] - d[2][2];
        v_ncp[0][2] = d[0][2] - d[0][1] - d[2][2] + d[2][1];
        v_ncp[0][3] = d[0][1] - d[0][3] - d[2][1] + d[2][3];
        
        v_ncp[1][0] = d[1][0] + d[2][0] - d[1][2] - d[2][2];
        v_ncp[1][1] = d[1][1] + d[1][2] + d[2][1] + d[2][2];
        v_ncp[1][2] = d[1][2] - d[1][1] + d[2][2] - d[2][1];
        v_ncp[1][3] = d[1][1] - d[1][3] + d[2][1] - d[2][3];
        
        v_ncp[2][0] = d[2][0] - d[1][0] - d[2][2] + d[1][2];
        v_ncp[2][1] = d[2][1] + d[2][2] - d[1][1] - d[1][2];
        v_ncp[2][2] = d[2][2] - d[2][1] - d[1][2] + d[1][1];
        v_ncp[2][3] = d[2][1] - d[2][3] - d[1][1] + d[1][3];
        
        v_ncp[3][0] = d[1][0] - d[1][2] - d[3][0] + d[3][2];
        v_ncp[3][1] = d[1][1] + d[1][2] - d[3][1] - d[3][2];
        v_ncp[3][2] = d[1][2] - d[1][1] - d[3][2] + d[3][1];
        v_ncp[3][3] = d[1][1] - d[1][3] - d[3][1] + d[3][3];

        // === 逐元素相乘并累加 ===
        #pragma unroll
        for (int i = 0; i < 4; ++i) {
            #pragma unroll
            for (int j = 0; j < 4; ++j) {
                m[i][j] += u_kc[i][j] * v_ncp[i][j];
            }
        }
    }

    // === 使用直接计算优化的输出变换 ===
    float Y[2][2];
    
    // 直接计算 A^T * m * A（避免中间临时数组temp_m）
    // Y = A^T * m * A，其中 A^T = [[1,1,1,0], [0,1,-1,-1]]
    Y[0][0] = m[0][0] + m[0][1] + m[0][2] + m[1][0] + m[1][1] + m[1][2] + m[2][0] + m[2][1] + m[2][2];
    Y[0][1] = m[0][1] - m[0][2] - m[0][3] + m[1][1] - m[1][2] - m[1][3] + m[2][1] - m[2][2] - m[2][3];
    Y[1][0] = m[1][0] + m[1][1] + m[1][2] - m[2][0] - m[2][1] - m[2][2] - m[3][0] - m[3][1] - m[3][2];
    Y[1][1] = m[1][1] - m[1][2] - m[1][3] - m[2][1] + m[2][2] + m[2][3] - m[3][1] + m[3][2] + m[3][3];

    // === 使用向量化访问优化的输出写入 ===
    int h0 = tile_y * 2;
    int w0 = tile_x * 2;
    int h1 = h0 + 1;
    int w1 = w0 + 1;
    
    // 在可能的情况下使用向量化写入
    if (h1 < outH && w1 < outW && outW % 2 == 0) {
        // 尝试使用float2向量化写入
        int out_base = (n * K + k) * outH * outW + h0 * outW + w0;
        if (w0 % 2 == 0) {
            float2* out_ptr = (float2*)output;
            // 写入行0
            out_ptr[out_base / 2] = make_float2(Y[0][0], Y[0][1]);
            // 写入行1
            out_ptr[(out_base + outW) / 2] = make_float2(Y[1][0], Y[1][1]);
        } else {
            // 回退到常规写入
            output[((n * K + k) * outH + h0) * outW + w0] = Y[0][0];
            output[((n * K + k) * outH + h0) * outW + w1] = Y[0][1];
            output[((n * K + k) * outH + h1) * outW + w0] = Y[1][0];
            output[((n * K + k) * outH + h1) * outW + w1] = Y[1][1];
        }
    } else {
        // 边界检查后备方案
        #pragma unroll
        for (int i = 0; i < 2; ++i) {
            #pragma unroll
            for (int j = 0; j < 2; ++j) {
                int h = tile_y * 2 + i;
                int w = tile_x * 2 + j;
                if (h < outH && w < outW) {
                    output[((n * K + k) * outH + h) * outW + w] = Y[i][j];
                }
            }
        }
    }
}

void winograd_conv(thrust::device_vector<float>& image,
                   thrust::device_vector<float>& filter, 
                   thrust::device_vector<float>& out,
                   thrust::device_vector<float>& U,
                   thrust::device_vector<float>& V, 
                   thrust::device_vector<float>& M,
                   int H, int W, int C, int K, int N) {
    const int outH = H - 2;
    const int outW = W - 2;
    
    // 为V100优化block大小 - 增加到512以获得更好的占用率
    const int threads_per_block = 512;
    int num_tiles = N * K * (outH / 2) * (outW / 2);
    int grid_size = (num_tiles + threads_per_block - 1) / threads_per_block;

    winograd_conv_kernel<<<grid_size, threads_per_block>>>(
        image.data().get(), filter.data().get(), out.data().get(),
        N, C, H, W, K, outH, outW
    );

    hipDeviceSynchronize();
}
